#include "mnist_cudnn/Layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

namespace CUDA_NETWORK
{
    /****************************************************************
     * Local Response Normalization definition                      *
    ****************************************************************/
    LRN::LRN(std::string name, unsigned n, double alpha, double beta, double k)
    {
        layerName = name;
        lrnN = n;
        lrnAlpha = alpha;
        lrnBeta = beta;
        lrnK = k;

        CheckCudnnErrors(hipdnnCreateLRNDescriptor(&normDesc));
        CheckCudnnErrors(hipdnnSetLRNDescriptor(normDesc, lrnN, lrnAlpha, lrnBeta, lrnK));
    }

    LRN::~LRN()
    {
        hipdnnDestroyLRNDescriptor(normDesc);
    }

    Blob<float> *LRN::Forward(Blob<float> *input)
    {
        // initilaize input and output
        if (input_ == nullptr || batchSize_ != input->num)
        {
            input_ = input;
            batchSize_ = input->num;
            inputDesc = input_->Tensor();

            if (output_ == nullptr)
                output_ = new Blob<float>(input->Shape());
            else
                output_->Reset(input->Shape());

            outputDesc = output_->Tensor();
        }
        
        CheckCudnnErrors(hipdnnLRNCrossChannelForward(cuda->Cudnn(),
                                                     normDesc,
                                                     HIPDNN_LRN_CROSS_CHANNEL,
                                                     &cuda->one,
                                                     inputDesc,
                                                     input_->Cuda(),
                                                     &cuda->zero,
                                                     outputDesc,
                                                     output_->Cuda()));
    #if (DEBUG_CONV & 0x01)
        input_->print(name_ + "::input", true, input_->n(), 28);
        output_->print(name_ + "::output", true);
    #endif

        return output_;
    }

    Blob<float> *LRN::Backward(Blob<float> *gradInput)
    {
        // initialize grad_output back-propagation space
        if (gradInput_ == nullptr || batchSize_ != gradInput->num)
        {
            gradOutput_ = gradInput;

            if (gradInput_ == nullptr)
                gradInput_ = new Blob<float>(input_->Shape());
            else
                gradInput_->Reset(input_->Shape());
        }

        CheckCudnnErrors(hipdnnLRNCrossChannelBackward(cuda->Cudnn(),
                                                      normDesc,
                                                      HIPDNN_LRN_CROSS_CHANNEL,
                                                      &cuda->one,
                                                      outputDesc,
                                                      output_->Cuda(),
                                                      outputDesc,
                                                      gradOutput_->Cuda(),
                                                      inputDesc,
                                                      input_->Cuda(),
                                                      &cuda->zero,
                                                      inputDesc,
                                                      gradInput_->Cuda()));

        return gradInput_;
    }
}