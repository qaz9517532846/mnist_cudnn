#include "hip/hip_runtime.h"
#include "mnist_cudnn/Layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

namespace CUDA_NETWORK
{
    /****************************************************************
    * Layer definition                                             *
    ****************************************************************/
    Layer::Layer()
    {

    }

    Layer::~Layer()
    {

    #if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
	    std::cout << "Destroy Layer: " << layerName << std::endl;
    #endif

        if(output_       != nullptr)  delete output_;
	    if(gradInput_    != nullptr)  delete gradInput_;

	    if(weights_      != nullptr)  delete weights_;
	    if(biases_       != nullptr)  delete biases_;
	    if(gradWeights_  != nullptr)  delete gradWeights_;
	    if(gradBiases_   != nullptr)  delete gradBiases_;
    }

    void Layer::InitWeightBias(unsigned int seed)
    {
	    CheckCudaErrors(hipDeviceSynchronize());

        if(weights_ == nullptr || biases_ == nullptr) return;

	    // Create random network
	    std::random_device rd;
	    std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

	    // He uniform distribution
	    float range = sqrt(6.f / input_->Size());	// He's initialization
	    std::uniform_real_distribution<> dis(-range, range);

	    for(int i = 0; i < weights_->Length(); i++)
		    weights_->Ptr()[i] = static_cast<float>(dis(gen));
	    for(int i = 0; i < biases_->Length(); i++)
		    biases_->Ptr()[i] = 0.f;

	    // copy initialized value to the device
	    weights_->To(DEV_TYPE::CUDA);
	    biases_->To(DEV_TYPE::CUDA);

	    std::cout << ".. initialized " << layerName << " layer .." << std::endl;
    }

    void Layer::UpdateWeightsBiases(float learningRate)
    {
	    float eps = -1.f * learningRate;

	    if(weights_ != nullptr && gradWeights_ != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    weights_->print(layerName + "::weights (before update)", true);
		    gradWeights_->print(layerName + "::gweights", true);
        #endif // DEBUG_UPDATE

		// w = w + eps * dw
		    CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), weights_->Length(), &eps, gradWeights_->Cuda(), 1, weights_->Cuda(), 1));

        #if(DEBUG_UPDATE)
		    weights->print(layerName + "weights (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }

	    if(biases_ != nullptr && gradBiases_ != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    biases_->print(layerName + "biases (before update)", true);
		    gradBiases_->print(layerName + "gbiases", true);
        #endif // DEBUG_UPDATE

		// b = b + eps * db
		CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), biases_->Length(), &eps, gradBiases_->Cuda(), 1, biases_->Cuda(), 1));

        #if (DEBUG_UPDATE)
		    biases_->print(layerName + "biases (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }
    }

    float Layer::GetLoss(Blob<float> *target)
    {
	    assert("No Loss layer has no loss." && false);
	    return EXIT_FAILURE;
    }

    int Layer::GetAccuracy(Blob<float> *target)
    {
	    assert("No Loss layer cannot estimate accuracy." && false);
	    return EXIT_FAILURE;
    }

    int Layer::LoadParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    // load weights and biases pretrained parameters
	    filenameWeights << layerName << ".bin";
	    if(weights_->FileRead(filenameWeights.str())) return -1;

	    filenameBiases << layerName << ".bias.bin";
	    if(biases_->FileRead(filenameBiases.str())) return -2;

	    std::cout << ".. loaded " << layerName << " pretrain parameter.." << std::endl;

	    return 0;
    }

    int Layer::SaveParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    std::cout << ".. saving " << layerName << " parameter ..";
	
	    // Write weights file
	    if(weights_)
	    {
		    filenameWeights << layerName << ".bin";
		    if(weights_->FileWrite(filenameWeights.str())) return -1;
	    }
	
	    // Write bias file
	    if(biases_)
	    {
		    filenameBiases << layerName << ".bias.bin";
		    if(biases_->FileWrite(filenameBiases.str())) return -2;
	    }

	    std::cout << " done .." << std::endl;

	    return 0;
    }

	std::string Layer::GetName()
	{
		return layerName;
	}

	void Layer::SetCudaContext(CudaContext *context)
	{
		cuda = context;
	}

	void Layer::SetLoadPretrain()
	{ 
		loadPretrain = true;
	}

    void Layer::SetGradientStop()
	{
		gradientStop = true;
	}

    void Layer::Freeze()
	{
		freeze_ = true;
	}
    
	void Layer::UnFreeze()
	{
		freeze_ = false;
	}

    /****************************************************************
    * Dense Layer                                                  *
    ****************************************************************/

	Dense::Dense(std::string name, int outSize)
	{
		layerName = name;
		outputSize = outSize;
	}

	Dense::~Dense()
	{
		if(dOneVec != nullptr) hipFree(dOneVec);
	}

	__global__ void InitOneVec(float* dOneVec, size_t length)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if (i >= length) return;

		dOneVec[i] = 1.f;
	}

	Blob<float> *Dense::Forward(Blob<float> *input)
	{
		// initialize weights and biases
		if(weights_ == nullptr)
		{
			// setup parameter size information
			inputSize  = input->channel * input->height * input->width;
		
			// initialize weight, bias, and output
			weights_ = new Blob<float>(1, 1, inputSize, outputSize);
			biases_  = new Blob<float>(1, 1, outputSize);
		}

		// initilaize input and output
		if(input_ == nullptr || batchSize_ != input_->num)
		{
			input_ = input;
			batchSize_  = input->num;

			if(output_ == nullptr)
				output_  = new Blob<float>(batchSize_, outputSize);
			else
				output_->Reset(batchSize_, outputSize);
		
			output_->Tensor();

			if(dOneVec != nullptr) hipFree(dOneVec);

			CheckCudaErrors(hipMalloc((void**)&dOneVec, sizeof(float) * batchSize_));
			InitOneVec<<<(batchSize_ + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D, BLOCK_DIM_1D >>>(dOneVec, batchSize_);

			// initialize weights and biases
			if(loadPretrain && freeze_)
			{
				if(LoadParameter())
				{
					std::cout << "error occurred.." << std::endl;
					exit(-1);
				}
			}
			else if(!freeze_)
			{
				InitWeightBias();
			}
			else
			{
				/* do nothing */
			}
		}


		// output = weights^T * input (without biases)
		CheckCublasErrors(
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_T, HIPBLAS_OP_N, 
				outputSize, batchSize_, inputSize,
				&cuda->one,  
				weights_->Cuda(), inputSize, 
				input_->Cuda(), inputSize,
				&cuda->zero, 
				output_->Cuda(),  outputSize));

		// output += biases * dOneVec ^ T
		CheckCublasErrors(
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_N, HIPBLAS_OP_N, 
				outputSize, batchSize_, 1,
				&cuda->one, 
				biases_->Cuda(), outputSize, 
				dOneVec, 1, 
				&cuda->one, 
				output_->Cuda(), outputSize));

	#if (DEBUG_DENSE & 0x01)
		input->Print(layerName + "::input",  true);
		weights->Print(layerName + "::weight", true);
		biases->Print(layerName + "::bias",   true);
		output->Print(layerName + "::output", true);
	#endif // DEBUG_DENSE

		return output_;
	}

	Blob<float> *Dense::Backward(Blob<float> *gradOutput)
	{
		if(gradWeights_ == nullptr)
		{
			gradWeights_ = new Blob<float>(weights_->Shape());
			gradBiases_  = new Blob<float>(biases_->Shape());
		}

		if(gradInput_ == nullptr || batchSize_ != gradOutput->num)
		{
			gradOutput_ = gradOutput;

			if (gradInput_ == nullptr)
				gradInput_   = new Blob<float>(input_->Shape());
			else
				gradInput_->Reset(input_->Shape());
		}

		// db = (dy) * d_one_vec
		hipblasSgemv(cuda->Cublas(),
				HIPBLAS_OP_N,
				outputSize, batchSize_,
				&cuda->one,
				gradOutput->Cuda(), outputSize,
				dOneVec, 1,
				&cuda->zero,
				gradBiases_->Cuda(), 1);

		// dw = x * (dy)^T
		hipblasSgemm(cuda->Cublas(),
			HIPBLAS_OP_N, HIPBLAS_OP_T,
			inputSize, outputSize, batchSize_,
			&cuda->one,
			input_->Cuda(),        inputSize,
			gradOutput->Cuda(),   outputSize,
			&cuda->zero,
			gradWeights_->Cuda(),  inputSize);

		// dx = W * dy
		if (!gradientStop)
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				inputSize, batchSize_, outputSize,
				&cuda->one,
				weights_->Cuda(),    inputSize,
				gradOutput->Cuda(), outputSize,
				&cuda->zero, 
				gradInput_->Cuda(),  inputSize);

	#if (DEBUG_DENSE & 0x02)
		std::cout << layerName << "[BACKWARD]" << std::endl;
		gradOutput->Print(layerName + "::gradients", true, gradOutput->num);
		gradWeights->Print(layerName + "::gfilter", true);
		gradBiases->Print(layerName + "::gbias", true);
		if(!gradientStop) gradInput->Print(layerName + "::gdata", true);
	#endif // DEBUG_DENSE

		return gradInput_;
	}

	/****************************************************************
 	* Activation Layer                                             *
 	****************************************************************/

	Activation::Activation(std::string name, hipdnnActivationMode_t mode, float coef)
	{
		layerName = name;
		actMode = mode;
		actCoef = coef;

		hipdnnCreateActivationDescriptor(&actDesc);
		hipdnnSetActivationDescriptor(actDesc, actMode, HIPDNN_PROPAGATE_NAN, actCoef);
	}

	Activation::~Activation()
	{
		hipdnnDestroyActivationDescriptor(actDesc);
	}

	Blob<float> *Activation::Forward(Blob<float> *input)
	{
		if(input == nullptr || batchSize_ != input->num)
		{
			input_ = input;
			inputDesc = input_->Tensor();
			batchSize_ = input_->num;

			if(output_ == nullptr)
				output_ = new Blob<float>(input_->Shape());
			else
				output_->Reset(input_->Shape());

			outputDesc = output_->Tensor();
		}

		hipdnnActivationForward(cuda->Cudnn(),
			actDesc,
			&cuda->one,
			inputDesc,
			input_->Cuda(),
			&cuda->zero,
			outputDesc,
			output_->Cuda());

		return output_;
	}

	Blob<float> *Activation::Backward(Blob<float> *gradOutput)
	{
		if (gradInput_ == nullptr || batchSize_ != gradOutput->num)
		{
			gradOutput = gradOutput;

			if (gradInput_ == nullptr)
				gradInput_ = new Blob<float>(input_->Shape());
			else
				gradInput_->Reset(input_->Shape());		
		}

		hipdnnActivationBackward(cuda->Cudnn(),
			actDesc,
			&cuda->one, 
			outputDesc, output_->Cuda(),
			outputDesc, gradOutput->Cuda(), 
			inputDesc, input_->Cuda(), 
			&cuda->zero, 
			inputDesc, gradInput_->Cuda());

		return gradInput_;
	}

	/****************************************************************
	 * Softmax definition                                           *
	 ****************************************************************/

	Softmax::Softmax(std::string name)
	{
		layerName = name;
	}

	Softmax::~Softmax()
	{

	}

	Blob<float> *Softmax::Forward(Blob<float> *input)
	{
		if(input == nullptr || batchSize_ != input->num)
		{
			input_ = input;
			inputDesc = input_->Tensor();
			batchSize_ = input_->num;
		
			if(output_ == nullptr)
				output_ = new Blob<float>(input_->Shape());
			else
				output_->Reset(input_->Shape());		

			outputDesc = output_->Tensor();
		}

	#if (DEBUG_SOFTMAX & 0x01)
		std::cout << layerName << "[FORWARD]" << std::endl;
		input_->Print(layerName + "::input", true, input_->num);
	#endif

		CheckCudnnErrors(
			hipdnnSoftmaxForward(cuda->Cudnn(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
				&cuda->one,  inputDesc,  input_->Cuda(),
				&cuda->zero, outputDesc, output_->Cuda()));

	#if (DEBUG_SOFTMAX & 0x01)
		output_->Print(layerName + "::output", true, input_->num);
	#endif

		return output_;
	}

	Blob<float> *Softmax::Backward(Blob<float> *target)
	{
		CheckCudaErrors(hipDeviceSynchronize());

		if(gradInput_ == nullptr || batchSize_ != target->num)
		{
			if (gradInput_ == nullptr)
				gradInput_ = new Blob<float>(input_->Shape());
			else
		 		gradInput_->Reset(input_->Shape());
		}

		// set grad_input_ as predict
		CheckCudaErrors(hipMemcpyAsync(gradInput_->Cuda(), 
			output_->Cuda(), output_->BufSize(), 
			hipMemcpyDeviceToDevice));

		// set gradInput = predict - target	
		CheckCublasErrors(
			hipblasSaxpy(cuda->Cublas(), target->Length(),
				&cuda->minusOne, target->Cuda(), 1,
				gradInput_->Cuda(), 1));

		// normalize the grad_output by the batch size
		int gradOutputSize = target->num * target->channel * target->height * target->width;
		float scale = 1.f / static_cast<float>(target->num);
		CheckCublasErrors(hipblasSscal(cuda->Cublas(), gradOutputSize, &scale, gradInput_->Cuda(), 1));

	#if (DEBUG_SOFTMAX & 0x02)
		std::cout << layerName << "[BACKWARD]" << std::endl;
		input_->Print( layerName + "::input", true);
		output_->Print(layerName + "::predict", true);
		target->Print( layerName + "::y", true, target->num);
		gradInput_->Print(layerName + "::dx", true, target->num);
	#endif

		return gradInput_;
	}

	float Softmax::GetLoss(Blob<float> *target)
	{
		return loss.Loss(output_, target);
	}

	int Softmax::GetAccuracy(Blob<float> *target)
	{
		int batchSize = output_->num;
		int outputSize = output_->Size();

		assert(batchSize == target->num);
		assert(outputSize == target->Size());

		float *hOutput, *hTarget;
		int idxOutput, idxTarget;
		int hitCount = 0;

		// get predicts and targets
		hOutput = output_->To(HOST);
		hTarget = target->To(HOST);

		// idxOutput = idxTarget = 0;
		for(int b = 0; b < batchSize; b++)
		{
			idxOutput = 0;
			idxTarget = 0;

			for (int i = 1; i < 10; i++)
			{
				if (hOutput[b * outputSize + i] > hOutput[b * outputSize + idxOutput])
					idxOutput = i;
				if (hTarget[b * outputSize + i] > hTarget[b * outputSize + idxTarget])
					idxTarget = i;
			}

			if (idxOutput == idxTarget)
				hitCount++;
		}

		return hitCount;
	}

	/****************************************************************
 	 * Layer definition                                             *
 	****************************************************************/

	/**
 	* Convolutional layer with bias
 	*/
	Conv2D::Conv2D(std::string name, int outChannels, int kernelSize, int stride, int padding, int dilation) :  outChannels(outChannels),
																												kernelSize(kernelSize),
																												stride(stride),
																												padding(padding),
																												dilation(dilation)
	{
		layerName = name;

		// create cudnn container handles
		hipdnnCreateFilterDescriptor(&filterDesc);
		hipdnnCreateConvolutionDescriptor(&convDesc);
		CheckCudnnErrors(
			hipdnnSetConvolution2dDescriptor(convDesc, padding, padding, stride,  stride, dilation, dilation, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

		dWorkspace = nullptr;
	}

	Conv2D::~Conv2D()
	{
		// distroy cudnn container resources
		hipdnnDestroyFilterDescriptor(filterDesc);
		hipdnnDestroyConvolutionDescriptor(convDesc);

		// terminate internal created blobs
		if(dWorkspace != nullptr)	hipFree(dWorkspace);
	}

	void Conv2D::SetWorkspace()
	{
		size_t tempSize = 0;

		hipdnnConvolutionFwdAlgoPerf_t			fwdAlgoPerfResults[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
		hipdnnConvolutionBwdFilterAlgoPerf_t 	bwdFilterAlgoPerfResults[HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT];
		hipdnnConvolutionBwdDataAlgoPerf_t		bwdDataAlgoPerfResults[HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM];

		// forward
	#if CUDNN_MAJOR >= 8
		int algoMaxCount;
		CheckCudnnErrors(cudnnGetConvolutionForwardAlgorithmMaxCount(cuda->Cudnn(), &algoMaxCount));
		std::cout << this->layerName << ": Available Algorithm Count [FWD]: " << algoMaxCount << std::endl;
		CheckCudnnErrors(cudnnGetConvolutionForwardAlgorithm_v7(cuda->Cudnn(), inputDesc, filterDesc, convDesc, outputDesc, algoMaxCount, 0, fwdAlgoPerfResults));
		convFwdAlgo = fwdAlgoPerfResults[0].algo;
	#else
		CheckCudnnErrors(hipdnnGetConvolutionForwardAlgorithm(cuda->Cudnn(), inputDesc, filterDesc, convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convFwdAlgo));
	#endif
		CheckCudnnErrors(hipdnnGetConvolutionForwardWorkspaceSize(cuda->Cudnn(), inputDesc, filterDesc, convDesc, outputDesc, convFwdAlgo, &tempSize));
		workspaceSize = std::max(workspaceSize, tempSize);

		// bwd - filter
	#if CUDNN_MAJOR >= 8
		CheckCudnnErrors(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(cuda->Cudnn(), &algoMaxCount));
		std::cout << this->layerName << ": Available Algorithm Count [BWD-filter]: " << algoMaxCount << std::endl;
		CheckCudnnErrors(cudnnGetConvolutionBackwardFilterAlgorithm_v7(cuda->Cudnn(), inputDesc, outputDesc, convDesc, filterDesc, algoMaxCount, 0, bwdFilterAlgoPerfResults));
		convBwdFilterAlgo = bwdFilterAlgoPerfResults[0].algo;
	#else
		CheckCudnnErrors(hipdnnGetConvolutionBackwardFilterAlgorithm(cuda->Cudnn(), inputDesc, outputDesc, convDesc, filterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &convBwdFilterAlgo));
	#endif
		CheckCudnnErrors(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cuda->Cudnn(),
			inputDesc, outputDesc, convDesc, filterDesc,
			convBwdFilterAlgo, &tempSize));
		workspaceSize = std::max(workspaceSize, tempSize);

		// bwd - data
	#if CUDNN_MAJOR >= 8
		CheckCudnnErrors(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(cuda->Cudnn(), &algoMaxCount));
		std::cout << this->layerName << ": Available Algorithm Count [BWD-data]: " << algoMaxCount << std::endl;
		CheckCudnnErrors(cudnnGetConvolutionBackwardDataAlgorithm_v7(cuda->Cudnn(), filterDesc, outputDesc, convDesc, inputDesc, algoMaxCount, 0, bwdDataAlgoPerfResults));
		convBwdDataAlgo = bwdDataAlgoPerfResults[0].algo;
	#else
		CheckCudnnErrors(hipdnnGetConvolutionBackwardDataAlgorithm(cuda->Cudnn(), filterDesc, outputDesc, convDesc, inputDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &convBwdDataAlgo));
	#endif
		CheckCudnnErrors(hipdnnGetConvolutionBackwardDataWorkspaceSize(cuda->Cudnn(), filterDesc, outputDesc, convDesc, inputDesc, convBwdDataAlgo, &tempSize));
		workspaceSize = std::max(workspaceSize, tempSize);

		if(workspaceSize > 0)
		{
			if(dWorkspace != nullptr) CheckCudaErrors(hipFree(dWorkspace));
			CheckCudaErrors(hipMalloc((void**)&dWorkspace, workspaceSize));
		}
	}

	Blob<float> *Conv2D::Forward(Blob<float> *input)
	{
		// initialize weights and bias
		if(weights_ == nullptr)
		{
			// initialize containers handles
			CheckCudnnErrors(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, outChannels, input->channel, kernelSize, kernelSize));
			weights_ = new Blob<float>(outChannels, input->channel, kernelSize, kernelSize);
			biases_  = new Blob<float>(1, outChannels);	// bias size
			biasDesc = biases_->Tensor();
		}
 
		// initilaize input and output
		if(input == nullptr || batchSize_ != input->num)
		{
			// initialize input
			input_ = input;
			inputDesc = input_->Tensor();
			batchSize_  = input_->num;

			// initilaize output
			CheckCudnnErrors(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &outputSize[0], &outputSize[1], &outputSize[2], &outputSize[3]));

			if (output_ == nullptr)
				output_  = new Blob<float>(outputSize);
			else
				output_->Reset(outputSize);

			outputDesc = output_->Tensor();

			// initialize workspace for cudnn
			SetWorkspace();

			// initialize weights
			if(loadPretrain && freeze_)
			{
				if(LoadParameter())
				{
					std::cout << "error occurred.." << std::endl;
					exit(-1);
				}
			}
			else if (!freeze_)
			{
				InitWeightBias();
			}
			else
			{
				/* do nothing */
			}
		}

		CheckCudnnErrors(hipdnnConvolutionForward(cuda->Cudnn(),
			&cuda->one,  inputDesc,  input_->Cuda(),
			filterDesc, weights_->Cuda(), convDesc, convFwdAlgo, dWorkspace,  workspaceSize,
			&cuda->zero, outputDesc, output_->Cuda()));

		CheckCudnnErrors(hipdnnAddTensor(cuda->Cudnn(), 
			&cuda->one, biasDesc, biases_->Cuda(), 
			&cuda->one, outputDesc, output_->Cuda()));

	#if (DEBUG_CONV & 0x01)
		input_->Print(layerName + "::input", true, input->num, 28);
		weights_->Print(layerName + "::weight", true);
		biases_->Print(layerName + "::bias", true);
		output_->Print(layerName + "::output", true);
	#endif

		return output_;
	}

	Blob<float> *Conv2D::Backward(Blob<float> *gradOutput)
	{
		// initialize gradOutput back-propagation space
		if(gradInput_ == nullptr || batchSize_ != gradOutput->num)
		{
			gradOutput_  = gradOutput;
			gradWeights_ = new Blob<float>(weights_->Shape());
			gradBiases_  = new Blob<float>(1, biases_->channel);

			if(gradInput_ == nullptr)
				gradInput_ = new Blob<float>(input_->Shape());
			else
				gradInput_->Reset(input_->Shape());
		}

		// gradients of biases
		CheckCudnnErrors(hipdnnConvolutionBackwardBias(cuda->Cudnn(), &cuda->one, outputDesc, gradOutput_->Cuda(), &cuda->zero, biasDesc, gradBiases_->Cuda()));
	
		// gradients of weights 
		CheckCudnnErrors(
			hipdnnConvolutionBackwardFilter(cuda->Cudnn(),
				&cuda->one, 
				inputDesc, input_->Cuda(), 
				outputDesc, gradOutput_->Cuda(),
				convDesc, convBwdFilterAlgo, dWorkspace, workspaceSize,
				&cuda->zero, 
				filterDesc, gradWeights_->Cuda()));

		// gradients of input data
		if (!gradientStop)
			CheckCudnnErrors(
				hipdnnConvolutionBackwardData(cuda->Cudnn(),
					&cuda->one, 
					filterDesc, weights_->Cuda(), 
					outputDesc, gradOutput_->Cuda(), 
					convDesc, convBwdDataAlgo, dWorkspace, workspaceSize,
					&cuda->zero, 
					inputDesc, gradInput_->Cuda()));

	#if (DEBUG_CONV & 0x02)
		std::cout << layerName << "[BACKWARD]" << std::endl;
		gradOutput_->Print(layerName + "::gradients", true);
		gradBiases_->Print(layerName + "gbias", true);
		gradWeights_->Print(layerName + "gfilter", true);
		if (!gradientStop)
			gradInput_->Print(layerName +"gdata", true);
	#endif

	#if (DEBUG_CONV & 0x04)
		gradOutput_->Print(layerName + "::gradients", true);
		gradBiases_->Print(layerName + "::gbias", true);
	#endif

		return gradInput_;
	}

	/****************************************************************
 	 * Layer definition                                             *
 	****************************************************************/
	Pooling::Pooling(std::string name, int kernelSize, int padding, int stride, hipdnnPoolingMode_t mode) :  poolKernelSize(kernelSize),
																											poolPadding(padding),
																											poolStride(stride),
																											poolMode(mode)
	{
		layerName = name;
		hipdnnCreatePoolingDescriptor(&poolDesc);
		hipdnnSetPooling2dDescriptor(poolDesc, poolMode, HIPDNN_PROPAGATE_NAN, poolKernelSize, poolKernelSize, poolPadding, poolPadding, poolStride, poolStride);
	}

	Pooling::~Pooling()
	{
		hipdnnDestroyPoolingDescriptor(poolDesc);
	}

	Blob<float> *Pooling::Forward(Blob<float> *input)
	{
		if(input == nullptr || batchSize_ != input->num)
		{
			input_ = input;

			// resource initialize
			inputDesc = input->Tensor();
			batchSize_ = input->num;
		
			// setting output
			hipdnnGetPooling2dForwardOutputDim(poolDesc, inputDesc, &outputSize[0], &outputSize[1], &outputSize[2], &outputSize[3]);
			if(output_ == nullptr)
				output_ = new Blob<float>(outputSize);
			else
				output_->Reset(outputSize);
		
			outputDesc = output_->Tensor();
		}

		hipdnnPoolingForward(cuda->Cudnn(), poolDesc,
			&cuda->one,   inputDesc,  input->Cuda(),
			&cuda->zero,  outputDesc, output_->Cuda());

		return output_;
	}

	Blob<float> *Pooling::Backward(Blob<float> *gradOutput)
	{
		if (gradInput_ == nullptr || batchSize_ != gradOutput->num)
		{
			gradOutput_ = gradOutput;

			if (gradInput_ == nullptr)
				gradInput_ = new Blob<float>(input_->Shape());
			else
				gradInput_->Reset(input_->Shape());
		}

		CheckCudnnErrors(
			hipdnnPoolingBackward(cuda->Cudnn(), poolDesc,
				&cuda->one,  
				outputDesc, output_->Cuda(),
				outputDesc, gradOutput_->Cuda(), 
				inputDesc,  input_->Cuda(), 
				&cuda->zero, 
				inputDesc,  gradInput_->Cuda()));

		return gradInput_;
	}
}