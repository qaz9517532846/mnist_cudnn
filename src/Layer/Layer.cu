#include "hip/hip_runtime.h"
#include "mnist_cudnn/Layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

namespace CUDA_NETWORK
{
    /****************************************************************
    * Layer definition                                             *
    ****************************************************************/
    Layer::Layer()
    {

    }

    Layer::~Layer()
    {

    #if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
	    std::cout << "Destroy Layer: " << name << std::endl;
    #endif

        if(output       != nullptr)  delete output;
	    if(gradInput    != nullptr)  delete gradInput;

	    if(weights      != nullptr)  delete weights;
	    if(biases       != nullptr)  delete biases;
	    if(gradWeights  != nullptr)  delete gradWeights;
	    if(gradBiases   != nullptr)  delete gradBiases;
    }

    void Layer::InitWeightBias(unsigned int seed)
    {
	    CheckCudaErrors(hipDeviceSynchronize());

        if(weights == nullptr || biases == nullptr) return;

	    // Create random network
	    std::random_device rd;
	    std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

	    // He uniform distribution
	    float range = sqrt(6.f / input->Size());	// He's initialization
	    std::uniform_real_distribution<> dis(-range, range);

	    for(int i = 0; i < weights->Length(); i++)
		    weights->Ptr()[i] = static_cast<float>(dis(gen));
	    for(int i = 0; i < biases->Length(); i++)
		    biases->Ptr()[i] = 0.f;

	    // copy initialized value to the device
	    weights->To(DEV_TYPE::CUDA);
	    biases->To(DEV_TYPE::CUDA);

	    std::cout << ".. initialized " << name << " layer .." << std::endl;
    }

    void Layer::UpdateWeightsBiases(float learningRate)
    {
	    float eps = -1.f * learningRate;

	    if(weights != nullptr && gradWeights != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    weights->print(name + "::weights (before update)", true);
		    gradWeights->print(name + "::gweights", true);
        #endif // DEBUG_UPDATE

		// w = w + eps * dw
		    CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), weights->Length(), &eps, gradWeights->Cuda(), 1, weights->Cuda(), 1));

        #if(DEBUG_UPDATE)
		    weights->print(name + "weights (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }

	    if(biases != nullptr && gradBiases != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    biases->print(name + "biases (before update)", true);
		    gradBiases->print(name + "gbiases", true);
        #endif // DEBUG_UPDATE

		// b = b + eps * db
		CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), biases->Length(), &eps, gradBiases->Cuda(), 1, biases->Cuda(), 1));

        #if (DEBUG_UPDATE)
		    biases->print(name + "biases (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }
    }

    float Layer::GetLoss(Blob<float> *target)
    {
	    assert("No Loss layer has no loss." && false);
	    return EXIT_FAILURE;
    }

    int Layer::GetAccuracy(Blob<float> *target)
    {
	    assert("No Loss layer cannot estimate accuracy." && false);
	    return EXIT_FAILURE;
    }

    int Layer::LoadParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    // load weights and biases pretrained parameters
	    filenameWeights << name << ".bin";
	    if(weights->FileRead(filenameWeights.str())) return -1;

	    filenameBiases << name << ".bias.bin";
	    if(biases->FileRead(filenameBiases.str())) return -2;

	    std::cout << ".. loaded " << name << " pretrain parameter.." << std::endl;

	    return 0;
    }

    int Layer::SaveParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    std::cout << ".. saving " << name << " parameter ..";
	
	    // Write weights file
	    if(weights)
	    {
		    filenameWeights << name << ".bin";
		    if(weights->FileWrite(filenameWeights.str())) return -1;
	    }
	
	    // Write bias file
	    if(biases)
	    {
		    filenameBiases << name << ".bias.bin";
		    if(biases->FileWrite(filenameBiases.str())) return -2;
	    }

	    std::cout << " done .." << std::endl;

	    return 0;
    }

    /****************************************************************
    * Dense Layer                                                  *
    ****************************************************************/

	Dense::Dense(std::string name, int outSize)
	{
		name = name;
		outputSize = outSize;
	}

	Dense::~Dense()
	{
		if(dOneVec != nullptr) hipFree(dOneVec);
	}

	__global__ void InitOneVec(float* dOneVec, size_t length)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if (i >= length) return;

		dOneVec[i] = 1.f;
	}

	Blob<float> *Dense::Forward(Blob<float> *input)
	{
		// initialize weights and biases
		if(weights == nullptr)
		{
			// setup parameter size information
			inputSize  = input->channel * input->height * input->width;
		
			// initialize weight, bias, and output
			weights = new Blob<float>(1, 1, inputSize, outputSize);
			biases  = new Blob<float>(1, 1, outputSize);
		}

		// initilaize input and output
		if(input == nullptr || batchSize != input->num)
		{
			input = input;
			batchSize  = input->num;

			if(output == nullptr)
				output  = new Blob<float>(batchSize, outputSize);
			else
				output->Reset(batchSize, outputSize);
		
			output->Tensor();

			if(dOneVec != nullptr) hipFree(dOneVec);

			CheckCudaErrors(hipMalloc((void**)&dOneVec, sizeof(float) * batchSize));
			InitOneVec<<<(batchSize + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D, BLOCK_DIM_1D >>>(dOneVec, batchSize);

			// initialize weights and biases
			if(loadPretrain && !freeze)
			{
				if(LoadParameter())
				{
					std::cout << "error occurred.." << std::endl;
					exit(-1);
				}
			}
			else if(!freeze)
			{
				InitWeightBias();
			}
			else
			{
				/* do nothing */
			}
		}


		// output = weights^T * input (without biases)
		CheckCublasErrors(
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_T, HIPBLAS_OP_N, 
				outputSize, batchSize, inputSize,
				&cuda->one,  
				weights->Cuda(), inputSize, 
				input->Cuda(), inputSize,
				&cuda->zero, 
				output->Cuda(),  outputSize));

		// output += biases * dOneVec ^ T
		CheckCublasErrors(
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_N, HIPBLAS_OP_N, 
				outputSize, batchSize, 1,
				&cuda->one, 
				biases->Cuda(), outputSize, 
				dOneVec, 1, 
				&cuda->one, 
				output->Cuda(), outputSize));

	#if (DEBUG_DENSE & 0x01)
		input->Print(name + "::input",  true);
		weights->Print(name + "::weight", true);
		biases->Print(name + "::bias",   true);
		output->Print(name + "::output", true);
	#endif // DEBUG_DENSE

		return output;
	}

	Blob<float> *Dense::Backward(Blob<float> *gradOutput)
	{
		if(gradWeights == nullptr)
		{
			gradWeights = new Blob<float>(weights->Shape());
			gradBiases  = new Blob<float>(biases->Shape());
		}

		if(gradInput == nullptr || batchSize != gradOutput->num)
		{
			gradOutput  = gradOutput;

			if (gradInput == nullptr)
				gradInput   = new Blob<float>(input->Shape());
			else
				gradInput->Reset(input->Shape());
		}

		// db = (dy) * d_one_vec
		hipblasSgemv(cuda->Cublas(),
				HIPBLAS_OP_N,
				outputSize, batchSize,
				&cuda->one,
				gradOutput->Cuda(), outputSize,
				dOneVec, 1,
				&cuda->zero,
				gradBiases->Cuda(), 1);

		// dw = x * (dy)^T
		hipblasSgemm(cuda->Cublas(),
			HIPBLAS_OP_N, HIPBLAS_OP_T,
			inputSize, outputSize, batchSize,
			&cuda->one,
			input->Cuda(),        inputSize,
			gradOutput->Cuda(),   outputSize,
			&cuda->zero,
			gradWeights->Cuda(),  inputSize);

		// dx = W * dy
		if (!gradientStop)
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				inputSize, batchSize, outputSize,
				&cuda->one,
				weights->Cuda(),    inputSize,
				gradOutput->Cuda(), outputSize,
				&cuda->zero, 
				gradInput->Cuda(),  inputSize);

	#if (DEBUG_DENSE & 0x02)
		std::cout << name << "[BACKWARD]" << std::endl;
		gradOutput->Print(name + "::gradients", true, gradOutput->num);
		gradWeights->Print(name + "::gfilter", true);
		gradBiases->Print(name + "::gbias", true);
		if(!gradientStop) gradInput->Print(name + "::gdata", true);
	#endif // DEBUG_DENSE

		return gradInput;
	}

	/****************************************************************
 	* Activation Layer                                             *
 	****************************************************************/


}