#include "hip/hip_runtime.h"
#include "mnist_cudnn/Layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

namespace CUDA_NETWORK
{    
	/****************************************************************
    * Layer definition                                             *
    ****************************************************************/
    Layer::Layer()
    {

    }

    Layer::~Layer()
    {

    #if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
	    std::cout << "Destroy Layer: " << layerName << std::endl;
    #endif

        if(output_       != nullptr)  delete output_;
	    if(gradInput_    != nullptr)  delete gradInput_;

	    if(weights_      != nullptr)  delete weights_;
	    if(biases_       != nullptr)  delete biases_;
	    if(gradWeights_  != nullptr)  delete gradWeights_;
	    if(gradBiases_   != nullptr)  delete gradBiases_;
    }

    void Layer::InitWeightBias(unsigned int seed)
    {
	    CheckCudaErrors(hipDeviceSynchronize());

        if(weights_ == nullptr || biases_ == nullptr) return;

	    // Create random network
	    std::random_device rd;
	    std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

	    // He uniform distribution
	    float range = sqrt(6.f / input_->Size());	// He's initialization
	    std::uniform_real_distribution<> dis(-range, range);

	    for(int i = 0; i < weights_->Length(); i++)
		    weights_->Ptr()[i] = static_cast<float>(dis(gen));
	    for(int i = 0; i < biases_->Length(); i++)
		    biases_->Ptr()[i] = 0.f;

		printf("He uniform distribution\n");

	    // copy initialized value to the device
	    weights_->To(DEV_TYPE::CUDA);
	    biases_->To(DEV_TYPE::CUDA);

	    std::cout << ".. initialized " << layerName << " layer .." << std::endl;
    }

    void Layer::UpdateWeightsBiases(float learningRate)
    {
	    float eps = -1.f * learningRate;

	    if(weights_ != nullptr && gradWeights_ != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    weights_->print(layerName + "::weights (before update)", true);
		    gradWeights_->print(layerName + "::gweights", true);
        #endif // DEBUG_UPDATE

		// w = w + eps * dw
		    CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), weights_->Length(), &eps, gradWeights_->Cuda(), 1, weights_->Cuda(), 1));

        #if(DEBUG_UPDATE)
		    weights->print(layerName + "weights (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }

	    if(biases_ != nullptr && gradBiases_ != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    biases_->print(layerName + "biases (before update)", true);
		    gradBiases_->print(layerName + "gbiases", true);
        #endif // DEBUG_UPDATE

		// b = b + eps * db
		CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), biases_->Length(), &eps, gradBiases_->Cuda(), 1, biases_->Cuda(), 1));

        #if (DEBUG_UPDATE)
		    biases_->print(layerName + "biases (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }
    }

    float Layer::GetLoss(Blob<float> *target)
    {
	    assert("No Loss layer has no loss." && false);
	    return EXIT_FAILURE;
    }

    int Layer::GetAccuracy(Blob<float> *target)
    {
	    assert("No Loss layer cannot estimate accuracy." && false);
	    return EXIT_FAILURE;
    }

    int Layer::LoadParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    // load weights and biases pretrained parameters
	    filenameWeights << layerName << ".bin";
	    if(weights_->FileRead(filenameWeights.str())) return -1;

	    filenameBiases << layerName << ".bias.bin";
	    if(biases_->FileRead(filenameBiases.str())) return -2;

	    std::cout << ".. loaded " << layerName << " pretrain parameter.." << std::endl;

	    return 0;
    }

    int Layer::SaveParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    std::cout << ".. saving " << layerName << " parameter ..";
	
	    // Write weights file
	    if(weights_)
	    {
		    filenameWeights << layerName << ".bin";
		    if(weights_->FileWrite(filenameWeights.str())) return -1;
	    }
	
	    // Write bias file
	    if(biases_)
	    {
		    filenameBiases << layerName << ".bias.bin";
		    if(biases_->FileWrite(filenameBiases.str())) return -2;
	    }

	    std::cout << " done .." << std::endl;

	    return 0;
    }

	std::string Layer::GetName()
	{
		return layerName;
	}

	void Layer::SetCudaContext(CudaContext *context)
	{
		cuda = context;
	}

	void Layer::SetLoadPretrain()
	{ 
		loadPretrain = true;
	}

    void Layer::SetGradientStop()
	{
		gradientStop = true;
	}

    void Layer::Freeze()
	{
		freeze_ = true;
	}
    
	void Layer::UnFreeze()
	{
		freeze_ = false;
	}
}