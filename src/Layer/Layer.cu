#include "hip/hip_runtime.h"
#include "mnist_cudnn/Layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

namespace CUDA_NETWORK
{
    /****************************************************************
    * Layer definition                                             *
    ****************************************************************/
    Layer::Layer()
    {

    }

    Layer::~Layer()
    {

    #if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
	    std::cout << "Destroy Layer: " << layerName << std::endl;
    #endif

        if(output       != nullptr)  delete output;
	    if(gradInput    != nullptr)  delete gradInput;

	    if(weights      != nullptr)  delete weights;
	    if(biases       != nullptr)  delete biases;
	    if(gradWeights  != nullptr)  delete gradWeights;
	    if(gradBiases   != nullptr)  delete gradBiases;
    }

    void Layer::InitWeightBias(unsigned int seed)
    {
	    CheckCudaErrors(hipDeviceSynchronize());

        if(weights == nullptr || biases == nullptr) return;

	    // Create random network
	    std::random_device rd;
	    std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

	    // He uniform distribution
	    float range = sqrt(6.f / input->Size());	// He's initialization
	    std::uniform_real_distribution<> dis(-range, range);

	    for(int i = 0; i < weights->Length(); i++)
		    weights->Ptr()[i] = static_cast<float>(dis(gen));
	    for(int i = 0; i < biases->Length(); i++)
		    biases->Ptr()[i] = 0.f;

	    // copy initialized value to the device
	    weights->To(DEV_TYPE::CUDA);
	    biases->To(DEV_TYPE::CUDA);

	    std::cout << ".. initialized " << layerName << " layer .." << std::endl;
    }

    void Layer::UpdateWeightsBiases(float learningRate)
    {
	    float eps = -1.f * learningRate;

	    if(weights != nullptr && gradWeights != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    weights->print(layerName + "::weights (before update)", true);
		    gradWeights->print(layerName + "::gweights", true);
        #endif // DEBUG_UPDATE

		// w = w + eps * dw
		    CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), weights->Length(), &eps, gradWeights->Cuda(), 1, weights->Cuda(), 1));

        #if(DEBUG_UPDATE)
		    weights->print(layerName + "weights (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }

	    if(biases != nullptr && gradBiases != nullptr)
	    {

        #if(DEBUG_UPDATE)
		    biases->print(layerName + "biases (before update)", true);
		    gradBiases->print(layerName + "gbiases", true);
        #endif // DEBUG_UPDATE

		// b = b + eps * db
		CheckCublasErrors(hipblasSaxpy(cuda->Cublas(), biases->Length(), &eps, gradBiases->Cuda(), 1, biases->Cuda(), 1));

        #if (DEBUG_UPDATE)
		    biases->print(layerName + "biases (after update)", true);
		    // getchar();
        #endif // DEBUG_UPDATE
	    }
    }

    float Layer::GetLoss(Blob<float> *target)
    {
	    assert("No Loss layer has no loss." && false);
	    return EXIT_FAILURE;
    }

    int Layer::GetAccuracy(Blob<float> *target)
    {
	    assert("No Loss layer cannot estimate accuracy." && false);
	    return EXIT_FAILURE;
    }

    int Layer::LoadParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    // load weights and biases pretrained parameters
	    filenameWeights << layerName << ".bin";
	    if(weights->FileRead(filenameWeights.str())) return -1;

	    filenameBiases << layerName << ".bias.bin";
	    if(biases->FileRead(filenameBiases.str())) return -2;

	    std::cout << ".. loaded " << layerName << " pretrain parameter.." << std::endl;

	    return 0;
    }

    int Layer::SaveParameter()
    {
	    std::stringstream filenameWeights, filenameBiases;

	    std::cout << ".. saving " << layerName << " parameter ..";
	
	    // Write weights file
	    if(weights)
	    {
		    filenameWeights << layerName << ".bin";
		    if(weights->FileWrite(filenameWeights.str())) return -1;
	    }
	
	    // Write bias file
	    if(biases)
	    {
		    filenameBiases << layerName << ".bias.bin";
		    if(biases->FileWrite(filenameBiases.str())) return -2;
	    }

	    std::cout << " done .." << std::endl;

	    return 0;
    }

	std::string Layer::GetName()
	{
		return layerName;
	}

	void Layer::SetCudaContext(CudaContext *context)
	{
		cuda = context;
	}

	void Layer::SetLoadPretrain()
	{ 
		loadPretrain = true;
	}

    void Layer::SetGradientStop()
	{
		gradientStop = true;
	}

    void Layer::Freeze()
	{
		freeze = true;
	}
    
	void Layer::UnFreeze()
	{
		freeze = false;
	}

    /****************************************************************
    * Dense Layer                                                  *
    ****************************************************************/

	Dense::Dense(std::string name, int outSize)
	{
		layerName = name;
		outputSize = outSize;
	}

	Dense::~Dense()
	{
		if(dOneVec != nullptr) hipFree(dOneVec);
	}

	__global__ void InitOneVec(float* dOneVec, size_t length)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if (i >= length) return;

		dOneVec[i] = 1.f;
	}

	Blob<float> *Dense::Forward(Blob<float> *input)
	{
		// initialize weights and biases
		if(weights == nullptr)
		{
			// setup parameter size information
			inputSize  = input->channel * input->height * input->width;
		
			// initialize weight, bias, and output
			weights = new Blob<float>(1, 1, inputSize, outputSize);
			biases  = new Blob<float>(1, 1, outputSize);
		}

		// initilaize input and output
		if(input == nullptr || batchSize != input->num)
		{
			input = input;
			batchSize  = input->num;

			if(output == nullptr)
				output  = new Blob<float>(batchSize, outputSize);
			else
				output->Reset(batchSize, outputSize);
		
			output->Tensor();

			if(dOneVec != nullptr) hipFree(dOneVec);

			CheckCudaErrors(hipMalloc((void**)&dOneVec, sizeof(float) * batchSize));
			InitOneVec<<<(batchSize + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D, BLOCK_DIM_1D >>>(dOneVec, batchSize);

			// initialize weights and biases
			if(loadPretrain && !freeze)
			{
				if(LoadParameter())
				{
					std::cout << "error occurred.." << std::endl;
					exit(-1);
				}
			}
			else if(!freeze)
			{
				InitWeightBias();
			}
			else
			{
				/* do nothing */
			}
		}


		// output = weights^T * input (without biases)
		CheckCublasErrors(
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_T, HIPBLAS_OP_N, 
				outputSize, batchSize, inputSize,
				&cuda->one,  
				weights->Cuda(), inputSize, 
				input->Cuda(), inputSize,
				&cuda->zero, 
				output->Cuda(),  outputSize));

		// output += biases * dOneVec ^ T
		CheckCublasErrors(
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_N, HIPBLAS_OP_N, 
				outputSize, batchSize, 1,
				&cuda->one, 
				biases->Cuda(), outputSize, 
				dOneVec, 1, 
				&cuda->one, 
				output->Cuda(), outputSize));

	#if (DEBUG_DENSE & 0x01)
		input->Print(layerName + "::input",  true);
		weights->Print(layerName + "::weight", true);
		biases->Print(layerName + "::bias",   true);
		output->Print(layerName + "::output", true);
	#endif // DEBUG_DENSE

		return output;
	}

	Blob<float> *Dense::Backward(Blob<float> *gradOutput)
	{
		if(gradWeights == nullptr)
		{
			gradWeights = new Blob<float>(weights->Shape());
			gradBiases  = new Blob<float>(biases->Shape());
		}

		if(gradInput == nullptr || batchSize != gradOutput->num)
		{
			gradOutput  = gradOutput;

			if (gradInput == nullptr)
				gradInput   = new Blob<float>(input->Shape());
			else
				gradInput->Reset(input->Shape());
		}

		// db = (dy) * d_one_vec
		hipblasSgemv(cuda->Cublas(),
				HIPBLAS_OP_N,
				outputSize, batchSize,
				&cuda->one,
				gradOutput->Cuda(), outputSize,
				dOneVec, 1,
				&cuda->zero,
				gradBiases->Cuda(), 1);

		// dw = x * (dy)^T
		hipblasSgemm(cuda->Cublas(),
			HIPBLAS_OP_N, HIPBLAS_OP_T,
			inputSize, outputSize, batchSize,
			&cuda->one,
			input->Cuda(),        inputSize,
			gradOutput->Cuda(),   outputSize,
			&cuda->zero,
			gradWeights->Cuda(),  inputSize);

		// dx = W * dy
		if (!gradientStop)
			hipblasSgemm(cuda->Cublas(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				inputSize, batchSize, outputSize,
				&cuda->one,
				weights->Cuda(),    inputSize,
				gradOutput->Cuda(), outputSize,
				&cuda->zero, 
				gradInput->Cuda(),  inputSize);

	#if (DEBUG_DENSE & 0x02)
		std::cout << layerName << "[BACKWARD]" << std::endl;
		gradOutput->Print(layerName + "::gradients", true, gradOutput->num);
		gradWeights->Print(layerName + "::gfilter", true);
		gradBiases->Print(layerName + "::gbias", true);
		if(!gradientStop) gradInput->Print(layerName + "::gdata", true);
	#endif // DEBUG_DENSE

		return gradInput;
	}

	/****************************************************************
 	* Activation Layer                                             *
 	****************************************************************/

	Activation::Activation(std::string name, hipdnnActivationMode_t mode, float coef)
	{
		layerName = name;
		actMode = mode;
		actCoef = coef;

		hipdnnCreateActivationDescriptor(&actDesc);
		hipdnnSetActivationDescriptor(actDesc, actMode, HIPDNN_PROPAGATE_NAN, actCoef);
	}

	Activation::~Activation()
	{
		hipdnnDestroyActivationDescriptor(actDesc);
	}

	Blob<float> *Activation::Forward(Blob<float> *input)
	{
		if(input == nullptr || batchSize != input->num)
		{
			input = input;
			inputDesc = input->Tensor();
			batchSize = input->num;

			if(output == nullptr)
				output = new Blob<float>(input->Shape());
			else
				output->Reset(input->Shape());

			outputDesc = output->Tensor();
		}

		hipdnnActivationForward(cuda->Cudnn(),
			actDesc,
			&cuda->one,
			inputDesc,
			input->Cuda(),
			&cuda->zero,
			outputDesc,
			output->Cuda());

		return output;
	}

	Blob<float> *Activation::Backward(Blob<float> *gradOutput)
	{
		if (gradInput == nullptr || batchSize != gradOutput->num)
		{
			gradOutput = gradOutput;

			if (gradInput == nullptr)
				gradInput = new Blob<float>(input->Shape());
			else
				gradInput->Reset(input->Shape());		
		}

		hipdnnActivationBackward(cuda->Cudnn(),
			actDesc,
			&cuda->one, 
			outputDesc, output->Cuda(),
			outputDesc, gradOutput->Cuda(), 
			inputDesc, input->Cuda(), 
			&cuda->zero, 
			inputDesc, gradInput->Cuda());

		return gradInput;
	}

	/****************************************************************
	 * Softmax definition                                           *
	 ****************************************************************/

	Softmax::Softmax(std::string name)
	{
		layerName = name;
	}

	Softmax::~Softmax()
	{

	}

	Blob<float> *Softmax::Forward(Blob<float> *input)
	{
		if(input == nullptr || batchSize != input->num)
		{
			input = input;
			inputDesc = input->Tensor();
			batchSize = input->num;
		
			if(output == nullptr)
				output = new Blob<float>(input->Shape());
			else
				output->Reset(input->Shape());		

			outputDesc = output->Tensor();
		}

	#if (DEBUG_SOFTMAX & 0x01)
		std::cout << layerName << "[FORWARD]" << std::endl;
		input->Print(layerName + "::input", true, input->num);
	#endif

		CheckCudnnErrors(
			hipdnnSoftmaxForward(cuda->Cudnn(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
				&cuda->one,  inputDesc,  input->Cuda(),
				&cuda->zero, outputDesc, output->Cuda()));

	#if (DEBUG_SOFTMAX & 0x01)
		output->Print(layerName + "::output", true, input->num);
	#endif

		return output;
	}

	Blob<float> *Softmax::Backward(Blob<float> *target)
	{
		CheckCudaErrors(hipDeviceSynchronize());

		if(gradInput == nullptr || batchSize != target->num)
		{
			if (gradInput == nullptr)
				gradInput = new Blob<float>(input->Shape());
			else
		 		gradInput->Reset(input->Shape());
		}

		// set grad_input_ as predict
		CheckCudaErrors(hipMemcpyAsync(gradInput->Cuda(), 
			output->Cuda(), output->BufSize(), 
			hipMemcpyDeviceToDevice));

		// set gradInput = predict - target	
		CheckCublasErrors(
			hipblasSaxpy(cuda->Cublas(), target->Length(),
				&cuda->minusOne, target->Cuda(), 1,
				gradInput->Cuda(), 1));

		// normalize the grad_output by the batch size
		int gradOutputSize = target->num * target->channel * target->height * target->width;
		float scale = 1.f / static_cast<float>(target->num);
		CheckCublasErrors(hipblasSscal(cuda->Cublas(), gradOutputSize, &scale, gradInput->Cuda(), 1));

	#if (DEBUG_SOFTMAX & 0x02)
		std::cout << layerName << "[BACKWARD]" << std::endl;
		input->Print( layerName + "::input", true);
		output->Print(layerName + "::predict", true);
		target->Print( layerName + "::y", true, target->num);
		gradInput->Print(layerName + "::dx", true, target->num);
	#endif

		return gradInput;
	}

	float Softmax::getLoss(Blob<float> *target)
	{
		return loss.Loss(output, target);
	}

	int Softmax::getAccuracy(Blob<float> *target)
	{
		int batchSize = output->num;
		int outputSize = output->Size();

		assert(batchSize == target->num);
		assert(outputSize == target->Size());

		float *hOutput, *hTarget;
		int idxOutput, idxTarget;
		int hitCount = 0;

		// get predicts and targets
		hOutput = output->To(HOST);
		hTarget = target->To(HOST);

		// idxOutput = idxTarget = 0;
		for(int b = 0; b < batchSize; b++)
		{
			idxOutput = 0;
			idxTarget = 0;

			for (int i = 1; i < 10; i++)
			{
				if (hOutput[b * outputSize + i] > hOutput[b * outputSize + idxOutput])
					idxOutput = i;
				if (hTarget[b * outputSize + i] > hTarget[b * outputSize + idxTarget])
					idxTarget = i;
			}

			if (idxOutput == idxTarget)
				hitCount++;
		}

		return hitCount;
	}

	/****************************************************************
 	 * Layer definition                                             *
 	****************************************************************/

	/**
 	* Convolutional layer with bias
 	*/
	Conv2D::Conv2D(std::string name, int outChannels, int kernelSize, int stride, int padding, int dilation) :  outChannels(outChannels),
																												kernelSize(kernelSize),
																												stride(stride),
																												padding(padding),
																												dilation(dilation)
	{
		layerName = name;

		// create cudnn container handles
		hipdnnCreateFilterDescriptor(&filterDesc);
		hipdnnCreateConvolutionDescriptor(&convDesc);
		CheckCudnnErrors(
			hipdnnSetConvolution2dDescriptor(convDesc, padding, padding, stride,  stride, dilation, dilation, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

		dWorkspace = nullptr;
	}

	Conv2D::~Conv2D()
	{
		// distroy cudnn container resources
		hipdnnDestroyFilterDescriptor(filterDesc);
		hipdnnDestroyConvolutionDescriptor(convDesc);

		// terminate internal created blobs
		if(dWorkspace != nullptr)	hipFree(dWorkspace);
	}

	void Conv2D::SetWorkspace()
	{
		size_t tempSize = 0;

		hipdnnConvolutionFwdAlgoPerf_t			fwdAlgoPerfResults[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
		hipdnnConvolutionBwdFilterAlgoPerf_t 	bwdFilterAlgoPerfResults[HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT];
		hipdnnConvolutionBwdDataAlgoPerf_t		bwdDataAlgoPerfResults[HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM];

		// forward
	#if CUDNN_MAJOR >= 8
		int algoMaxCount;
		CheckCudnnErrors(cudnnGetConvolutionForwardAlgorithmMaxCount(cuda->Cudnn(), &algoMaxCount));
		std::cout << this->layerName << ": Available Algorithm Count [FWD]: " << algoMaxCount << std::endl;
		CheckCudnnErrors(cudnnGetConvolutionForwardAlgorithm_v7(cuda->Cudnn(), inputDesc, filterDesc, convDesc, outputDesc, algoMaxCount, 0, fwdAlgoPerfResults));
		convFwdAlgo = fwdAlgoPerfResults[0].algo;
	#else
		CheckCudnnErrors(hipdnnGetConvolutionForwardAlgorithm(cuda->Cudnn(), inputDesc, filterDesc, convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convFwdAlgo));
	#endif
		CheckCudnnErrors(hipdnnGetConvolutionForwardWorkspaceSize(cuda->Cudnn(), inputDesc, filterDesc, convDesc, outputDesc, convFwdAlgo, &tempSize));
		workspaceSize = std::max(workspaceSize, tempSize);

		// bwd - filter
	#if CUDNN_MAJOR >= 8
		CheckCudnnErrors(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(cuda->Cudnn(), &algoMaxCount));
		std::cout << this->layerName << ": Available Algorithm Count [BWD-filter]: " << algoMaxCount << std::endl;
		CheckCudnnErrors(cudnnGetConvolutionBackwardFilterAlgorithm_v7(cuda->Cudnn(), inputDesc, outputDesc, convDesc, filterDesc, algoMaxCount, 0, bwdFilterAlgoPerfResults));
		convBwdFilterAlgo = bwdFilterAlgoPerfResults[0].algo;
	#else
		CheckCudnnErrors(hipdnnGetConvolutionBackwardFilterAlgorithm(cuda->Cudnn(), inputDesc, outputDesc, convDesc, filterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &convBwdFilterAlgo));
	#endif
		CheckCudnnErrors(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cuda->Cudnn(),
			inputDesc, outputDesc, convDesc, filterDesc,
			convBwdFilterAlgo, &tempSize));
		workspaceSize = std::max(workspaceSize, tempSize);

		// bwd - data
	#if CUDNN_MAJOR >= 8
		CheckCudnnErrors(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(cuda->Cudnn(), &algoMaxCount));
		std::cout << this->layerName << ": Available Algorithm Count [BWD-data]: " << algoMaxCount << std::endl;
		CheckCudnnErrors(cudnnGetConvolutionBackwardDataAlgorithm_v7(cuda->Cudnn(), filterDesc, outputDesc, convDesc, inputDesc, algoMaxCount, 0, bwdDataAlgoPerfResults));
		convBwdDataAlgo = bwdDataAlgoPerfResults[0].algo;
	#else
		CheckCudnnErrors(hipdnnGetConvolutionBackwardDataAlgorithm(cuda->Cudnn(), filterDesc, outputDesc, convDesc, inputDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &convBwdDataAlgo));
	#endif
		CheckCudnnErrors(hipdnnGetConvolutionBackwardDataWorkspaceSize(cuda->Cudnn(), filterDesc, outputDesc, convDesc, inputDesc, convBwdDataAlgo, &tempSize));
		workspaceSize = std::max(workspaceSize, tempSize);

		if(workspaceSize > 0)
		{
			if(dWorkspace != nullptr) CheckCudaErrors(hipFree(dWorkspace));
			CheckCudaErrors(hipMalloc((void**)&dWorkspace, workspaceSize));
		}
	}

	Blob<float> *Conv2D::Forward(Blob<float> *input)
	{
		// initialize weights and bias
		if(weights == nullptr)
		{
			// initialize containers handles
			CheckCudnnErrors(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, outChannels, input->channel, kernelSize, kernelSize));
			weights = new Blob<float>(outChannels, input->channel, kernelSize, kernelSize);
			biases  = new Blob<float>(1, outChannels);	// bias size
			biasDesc = biases->Tensor();
		}
 
		// initilaize input and output
		if(input == nullptr || batchSize != input->num)
		{
			// initialize input
			input = input;
			inputDesc = input->Tensor();
			batchSize  = input->num;

			printf("batchSize = %d, Size 0 = %d, Size 1 = %d, Size 2 = %d, Size 3 = %d\n", batchSize, outputSize[0], outputSize[1], outputSize[2], outputSize[3]);

			// initilaize output
			CheckCudnnErrors(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &outputSize[0], &outputSize[1], &outputSize[2], &outputSize[3]));

			if (output == nullptr)
				output  = new Blob<float>(outputSize);
			else
				output->Reset(outputSize);

			outputDesc = output->Tensor();

			// initialize workspace for cudnn
			SetWorkspace();

			// initialize weights
			if(loadPretrain && !freeze)
			{
				if(LoadParameter())
				{
					std::cout << "error occurred.." << std::endl;
					exit(-1);
				}
			}
			else if (!freeze)
			{
				InitWeightBias();
			}
			else
			{
				/* do nothing */
			}
		}

		CheckCudnnErrors(hipdnnConvolutionForward(cuda->Cudnn(),
			&cuda->one,  inputDesc,  input->Cuda(),
			filterDesc, weights->Cuda(), convDesc, convFwdAlgo, dWorkspace,  workspaceSize,
			&cuda->zero, outputDesc, output->Cuda()));

		CheckCudnnErrors(hipdnnAddTensor(cuda->Cudnn(), 
			&cuda->one, biasDesc, biases->Cuda(), 
			&cuda->one, outputDesc, output->Cuda()));

	#if (DEBUG_CONV & 0x01)
		input->Print(layerName + "::input", true, input->num, 28);
		weights->Print(layerName + "::weight", true);
		biases->Print(layerName + "::bias", true);
		output->Print(layerName + "::output", true);
	#endif

		return output;
	}

	Blob<float> *Conv2D::Backward(Blob<float> *gradOutput)
	{
		// initialize gradOutput back-propagation space
		if(gradInput == nullptr || batchSize != gradOutput->num)
		{
			gradOutput  = gradOutput;
			gradWeights = new Blob<float>(weights->Shape());
			gradBiases  = new Blob<float>(1, biases->channel);

			if(gradInput == nullptr)
				gradInput = new Blob<float>(input->Shape());
			else
				gradInput->Reset(input->Shape());
		}

		// gradients of biases
		CheckCudnnErrors(hipdnnConvolutionBackwardBias(cuda->Cudnn(), &cuda->one, outputDesc, gradOutput->Cuda(), &cuda->zero, biasDesc, gradBiases->Cuda()));
	
		// gradients of weights 
		CheckCudnnErrors(
			hipdnnConvolutionBackwardFilter(cuda->Cudnn(),
				&cuda->one, 
				inputDesc, input->Cuda(), 
				outputDesc, gradOutput->Cuda(),
				convDesc, convBwdFilterAlgo, dWorkspace, workspaceSize,
				&cuda->zero, 
				filterDesc, gradWeights->Cuda()));

		// gradients of input data
		if (!gradientStop)
			CheckCudnnErrors(
				hipdnnConvolutionBackwardData(cuda->Cudnn(),
					&cuda->one, 
					filterDesc, weights->Cuda(), 
					outputDesc, gradOutput->Cuda(), 
					convDesc, convBwdDataAlgo, dWorkspace, workspaceSize,
					&cuda->zero, 
					inputDesc, gradInput->Cuda()));

	#if (DEBUG_CONV & 0x02)
		std::cout << layerName << "[BACKWARD]" << std::endl;
		gradOutput->Print(layerName + "::gradients", true);
		gradBiases->Print(layerName + "gbias", true);
		gradWeights->Print(layerName + "gfilter", true);
		if (!gradientStop)
			gradInput->Print(layerName +"gdata", true);
	#endif

	#if (DEBUG_CONV & 0x04)
		gradOutput->Print(layerName + "::gradients", true);
		gradBiases->Print(layerName + "::gbias", true);
	#endif

		return gradInput;
	}

	/****************************************************************
 	 * Layer definition                                             *
 	****************************************************************/
	Pooling::Pooling(std::string name, int kernelSize, int padding, int stride, hipdnnPoolingMode_t mode) :  poolKernelSize(kernelSize),
																											poolPadding(padding),
																											poolStride(stride),
																											poolMode(mode)
	{
		layerName = name;
		hipdnnCreatePoolingDescriptor(&poolDesc);
		hipdnnSetPooling2dDescriptor(poolDesc, poolMode, HIPDNN_PROPAGATE_NAN, poolKernelSize, poolKernelSize, poolPadding, poolPadding, poolStride, poolStride);
	}

	Pooling::~Pooling()
	{
		hipdnnDestroyPoolingDescriptor(poolDesc);
	}

	Blob<float> *Pooling::Forward(Blob<float> *input)
	{
		if(input == nullptr || batchSize != input->num)
		{
			input = input;

			// resource initialize
			inputDesc = input->Tensor();
			batchSize = input->num;
		
			// setting output
			hipdnnGetPooling2dForwardOutputDim(poolDesc, inputDesc, &outputSize[0], &outputSize[1], &outputSize[2], &outputSize[3]);
			if(output == nullptr)
				output = new Blob<float>(outputSize);
			else
				output->Reset(outputSize);
		
			outputDesc = output->Tensor();
		}

		hipdnnPoolingForward(cuda->Cudnn(), poolDesc,
			&cuda->one,   inputDesc,  input->Cuda(),
			&cuda->zero,  outputDesc, output->Cuda());

		return output;
	}

	Blob<float> *Pooling::Backward(Blob<float> *gradOutput)
	{
		if (gradInput == nullptr || batchSize != gradOutput->num)
		{
			gradOutput = gradOutput;

			if (gradInput == nullptr)
				gradInput = new Blob<float>(input->Shape());
			else
				gradInput->Reset(input->Shape());
		}

		CheckCudnnErrors(
			hipdnnPoolingBackward(cuda->Cudnn(), poolDesc,
				&cuda->one,  
				outputDesc, output->Cuda(),
				outputDesc, gradOutput->Cuda(), 
				inputDesc,  input->Cuda(), 
				&cuda->zero, 
				inputDesc,  gradInput->Cuda()));

		return gradInput;
	}
}